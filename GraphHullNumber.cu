#include "hip/hip_runtime.h"
#include "UndirectedSparseGraph.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DEFAULT_THREAD_PER_BLOCK 32
#define MAX_DEFAULT_SIZE_QUEUE 50
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))



//References and Examples:
//https://msdn.microsoft.com/en-us/library/aa289166(v=vs.71).aspx
//D. Knuth. The Art of Computer Programming: Generating All Combinations and Partitions. Number v. 3-4 in Art of Computer Programming. Addison-Wesley, 2005.

__host__ __device__
int maxCombinations(int n, int k) {
    if (n == 0 || k == 0) {
        return 0;
    }
    if (n < k) {
        return 0;
    }
    if (n == k) {
        return 1;
    }
    long delta, idxMax;
    if (k < n - k) {
        delta = n - k;
        idxMax = k;
    } else {
        delta = k;
        idxMax = n - k;
    }

    long ans = delta + 1;
    for (long i = 2; i <= idxMax; ++i) {
        ans = (ans * (delta + i)) / i;
    }
    return ans;
}

__host__ __device__
void initialCombination(int n, int k, int* combinationArray, int idx) {
    int a = n;
    int b = k;
    long x = (maxCombinations(n, k) - 1) - idx;
    for (int i = 0; i < k; ++i) {
        combinationArray[i] = a - 1;
        while (maxCombinations(combinationArray[i], b) > x) {
            --combinationArray[i];
        }
        x = x - maxCombinations(combinationArray[i], b);
        a = combinationArray[i];
        b = b - 1;
    }

    for (int i = 0; i < k; ++i) {
        combinationArray[i] = (n - 1) - combinationArray[i];
    }
}

void initialCombination(int n, int k, int* combinationArray) {
    for (int i = 0; i < k; i++) {
        combinationArray[i] = i;
    }
}

__host__ __device__
void nextCombination(int n,
        int k,
        int* currentCombination) {
    if (currentCombination[0] == n - k) {
        return;
    }
    int i;
    for (i = k - 1; i > 0 && currentCombination[i] == n - k + i; --i);
    ++currentCombination[i];
    for (int j = i; j < k - 1; ++j) {
        currentCombination[j + 1] = currentCombination[j] + 1;
    }
}

__host__ __device__ void printCombination(int *currentCombination,
        int sizeComb) {
    printf("Combination: {");
    for (int i = 0; i < sizeComb; i++) {
        printf("%d", currentCombination[i]);
        if (i < sizeComb - 1) {
            printf(", ");
        }
    }
    printf("}");
}

bool verboseSerial = false;
bool verboseParallel = false;

__host__ __device__
void printQueue(int *queue, int headQueue, int tailQueue) {
    printf("\nQueue(%d):{", tailQueue - headQueue);
    for (int i = headQueue; i <= tailQueue; i++) {
        printf("%d", queue[i]);
        if (i < tailQueue) {
            printf(", ");
        }
    }
    printf("}\n");
}

__host__ __device__
int checkConvexityP3CSR(int *csrColIdxs, int nvertices,
        int *csrRowOffset, int sizeRowOffset,
        unsigned char *aux,
        int auxSize,
        int *currentCombination,
        int sizeComb) {
    //clean aux vector            
    for (int i = 0; i < auxSize; i++) {
        aux[i] = 0;
    }
    int closeCount = 0;
    int maxSizeQueue = MAX(auxSize / 2, MAX_DEFAULT_SIZE_QUEUE);
    int *queue = (int *) malloc(maxSizeQueue * sizeof (int));
    int headQueue = 0;
    int tailQueue = -1;

    for (int i = 0; i < sizeComb; i++) {
        tailQueue = (tailQueue + 1) % maxSizeQueue;
        queue[tailQueue] = currentCombination[i];
    }

    //    if (verboseSerial) {
    //        printCombination(currentCombination, sizeComb);
    //    }

    while (headQueue <= tailQueue) {
        //                if (verboseSerial) {
        //                    printQueue(queue, headQueue, tailQueue);
        //                }
        int verti = queue[headQueue];
        headQueue = (headQueue + 1) % maxSizeQueue;
        //        if (verboseSerial) {
        //            printf("vi: %d", verti);
        //        }

        if (aux[verti] < PROCESSED && verti < nvertices) {
            closeCount++;
            int end = csrColIdxs[verti + 1];
            for (int i = csrColIdxs[verti]; i < end; i++) {
                int vertn = csrRowOffset[i];
                if (vertn != verti) {
                    unsigned char previousValue = aux[vertn];
                    if (previousValue < INCLUDED) {
                        aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                    }
                    if (previousValue < INCLUDED && aux[vertn] >= INCLUDED) {
                        tailQueue = (tailQueue + 1) % maxSizeQueue;
                        queue[tailQueue] = vertn;
                        //                        if (verboseSerial) printf("\n\t vert-includ: %d, ", vertn);
                    }
                }
            }
            aux[verti] = PROCESSED;
        }
    }
    free(queue);
    return closeCount;
}

int checkConvexityP3(UndirectedCSRGraph *graph,
        unsigned char *aux,
        int auxSize,
        int *currentCombination,
        int sizeComb) {
    return checkConvexityP3CSR(graph->getCsrColIdxs(), graph->getVerticesCount(),
            graph->getCsrRowOffset(), graph->getSizeRowOffset(),
            aux, auxSize, currentCombination, sizeComb);

    //    int checkConvexityP3CSR(int *csrColIdxs, int nvertices,
    //        int *csrRowOffset, int sizeRowOffset,
    //        unsigned char *aux,
    //        int auxSize,
    //        int *currentCombination,
    //        int sizeComb)
}

void serialFindHullNumber(UndirectedCSRGraph *graph) {
    int nvs = graph->getVerticesCount();
    int k;
    unsigned char *aux = new unsigned char [nvs];
    int *currentCombination;

    int currentSize = 0;
    int maxSize = nvs;
    int sizeCurrentHcp3 = 0;

    bool found = false;

    graph->begin_serial_time = clock();
    while (currentSize < maxSize && !found) {
        currentSize++;
        k = currentSize;
        int maxCombination = maxCombinations(nvs, k);
        currentCombination = (int *) malloc(k * sizeof (int));
        initialCombination(nvs, k, currentCombination);
        if (verboseSerial)
            printf("\nComb(%d,%d)=%d", nvs, k, maxCombination);
        for (int i = 0; i < maxCombination && !found; i++) {
            sizeCurrentHcp3 = checkConvexityP3(graph, aux, nvs, currentCombination, k);
            //                        printCombination(currentCombination, k);
            found = (sizeCurrentHcp3 == nvs);
            if (!found)
                nextCombination(nvs, k, currentCombination);
        }
        if (found) {
            printf("Result Serial: ");
            printCombination(currentCombination, currentSize);
            printf(" |S| = %d |hcp3(S)| = |V(g)| = %d\n", k, sizeCurrentHcp3);
        }
        free(currentCombination);
    }
    graph->end_serial_time = clock();
    free(aux);
}

__global__ void kernelFindHullNumber(int *csrColIdxs, int nvertices,
        int *csrRowOffset, int sizeRowOffset, int maxCombination,
        int k, int offset, int *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //    printf("\nT(%2d)= G(%d,%d),C(%d,%d)=%d", idx, nvertices, sizeRowOffset, nvertices, k, maxCombination);
    //    int idx = 1;
    bool found = false;
    int *currentCombination = (int *) malloc(k * sizeof (int));
    unsigned char *aux = (unsigned char *) malloc(nvertices * sizeof (unsigned char));
    int sizeCurrentHcp3 = 0;
    int limmit = (idx + 1) * offset;
    if (limmit > maxCombination) {
        limmit = maxCombination;
    }
    int i = idx * offset;

    int maxSizeQueue = MAX(nvertices / 2, MAX_DEFAULT_SIZE_QUEUE);
    int *queue = (int *) malloc(maxSizeQueue * sizeof (int));

    //    printf("\nT(%2d:|%d|-|%d|)", idx, i, limmit);

    //    printf("\nN. vertices: %d", nvertices);
    //    printf("\nSize RowOffset: %d", sizeRowOffset);
    //    printf("\nR: {");
    //    for (int i = 0; i < nvertices + 1; i++) {
    //        printf("%d", csrColIdxs[i]);
    //        if (i < nvertices) {
    //            printf(", ");
    //        }
    //    }
    //    printf("} \nC: {");
    //    for (int i = 0; i < sizeRowOffset; i++) {
    //        printf("%d", csrRowOffset[i]);
    //        if (i < sizeRowOffset - 1) {
    //            printf(", ");
    //        }
    //    }
    //    printf("}");

    initialCombination(nvertices, k, currentCombination, i);
    //    printf("\nT(%2d:|%d|-|%d|): ", idx, i, limmit);
    //    printCombination(currentCombination, k);
    while (i < limmit && !found && !result[0]) {
        //        sizeCurrentHcp3 = checkConvexityP3CSR(csrColIdxs, nvertices,
        //                csrRowOffset, sizeRowOffset,
        //                aux, nvertices, currentCombination, k);

        int headQueue = 0;
        int tailQueue = -1;
        sizeCurrentHcp3 = 0;

        for (int y = 0; y < nvertices; y++) {
            aux[y] = 0;
        }

        for (int j = 0; j < k; j++) {
            tailQueue = (tailQueue + 1) % maxSizeQueue;
            queue[tailQueue] = currentCombination[j];
        }
        //        printCombination(currentCombination, k);
        //        printf("\n");
        while (headQueue <= tailQueue) {
            //            printQueue(queue, headQueue, tailQueue);

            int verti = queue[headQueue];
            headQueue = (headQueue + 1) % maxSizeQueue;

            //            printf("vi: %d", verti);

            if (aux[verti] < PROCESSED) {
                sizeCurrentHcp3++;
                int end = csrColIdxs[verti + 1];
                int x = csrColIdxs[verti];
                //                printf("vi: %d -- processando d(vi[%d-%d])=%d", verti, x, end, end - csrColIdxs[verti]);
                for (; x < end; x++) {
                    int vertn = csrRowOffset[x];
                    if (vertn != verti) {
                        unsigned char previousValue = aux[vertn];
                        if (previousValue < INCLUDED) {
                            aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                        }
                        if (previousValue < INCLUDED && aux[vertn] >= INCLUDED) {
                            tailQueue = (tailQueue + 1) % maxSizeQueue;
                            queue[tailQueue] = vertn;
                        }
                    }
                }
                aux[verti] = PROCESSED;
            }
            //            else {
            //                printf("vi: %d -- ja processado", verti);
            //            }
        }

        //        printf("\nHcp3(idx-%d,il-%d,k-%d)=%d ", idx, i, k, sizeCurrentHcp3);
        found = (sizeCurrentHcp3 == nvertices);
        if (!found)
            nextCombination(nvertices, k, currentCombination);
        i++;
    }
    if (found) {
        result[0] = sizeCurrentHcp3;
        result[1] = (i - 1);
        //        printf("\n\nFind Parallel\n");
    }
    free(queue);
    free(currentCombination);
    free(aux);
}

void parallelFindHullNumber(UndirectedCSRGraph *graph) {
    int nvs = graph->getVerticesCount();
    int k;
    int currentSize = 0;
    int maxSize = nvs;
    int result[2];
    result[0] = result[1] = 0;
    int* csrColIdxs = graph->getCsrColIdxs();
    int verticesCount = graph->getVerticesCount();
    int* csrRowOffset = graph->getCsrRowOffset();
    int sizeRowOffset = graph->getSizeRowOffset();

    int* csrColIdxsGpu;
    int* csrRowOffsetGpu;
    int *resultGpu;

    int numBytesClsIdx = sizeof (int)*(verticesCount + 1);
    hipMalloc((void**) &csrColIdxsGpu, numBytesClsIdx);

    int numBytesRowOff = sizeof (int)*sizeRowOffset;
    hipMalloc((void**) &csrRowOffsetGpu, numBytesRowOff);

    int numBytesResult = sizeof (int)*2;
    hipMalloc((void**) &resultGpu, numBytesResult);

    if (resultGpu == NULL || csrRowOffsetGpu == NULL || csrColIdxsGpu == NULL) {
        perror("Failed allocate memory in GPU");
    }

    hipError_t r = hipMemcpy(csrColIdxsGpu, csrColIdxs, numBytesClsIdx, hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        perror("Failed to copy memory");
    }
    r = hipMemcpy(csrRowOffsetGpu, csrRowOffset, numBytesRowOff, hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        perror("Failed to copy memory");
    }
    r = hipMemcpy(resultGpu, result, numBytesResult, hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        perror("Failed to copy memory");
    }

    bool found = false;

    graph->begin_parallel_time = clock();
    //    while (currentSize < maxSize - 1 && !found) {
    while (currentSize < 15 && !found) {
        currentSize++;
        k = currentSize;
        int maxCombination = maxCombinations(nvs, k);

        int threadsPerBlock = DEFAULT_THREAD_PER_BLOCK;
        if (DEFAULT_THREAD_PER_BLOCK > maxCombination) {
            threadsPerBlock = maxCombination / 3;
        }
        //        int threadsPerBlock = 4;
        printf("\n%d-Comb(%d,%d)=%d", k, verticesCount, k, maxCombination);
        int offset = maxCombination / threadsPerBlock;
        //        if ((maxCombination % threadsPerBlock) > 0) {
        //            offset++;
        //        }
        kernelFindHullNumber << < 1, threadsPerBlock >>> (csrColIdxsGpu, verticesCount,
                csrRowOffsetGpu, sizeRowOffset, maxCombination, k, offset, resultGpu);
        hipMemcpy(result, resultGpu, numBytesResult, hipMemcpyDeviceToHost);
        found = (result[0] == nvs);
    }
    if (found) {
        printf("Result Parallel: S=%d-Comb(%d,%d) |S| = %d |Hcp3(S)| = |V(g)| = %d\n", result[1], nvs, k, k, result[0]);
    }
    //    else {
    //        printf("Result Parallel: Not found!");
    //    }
    graph->end_parallel_time = clock();
    hipFree(resultGpu);
    hipFree(csrRowOffsetGpu);
    hipFree(csrColIdxsGpu);
}
